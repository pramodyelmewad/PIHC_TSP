
#include <hip/hip_runtime.h>
#include"stdio.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include"math.h"
#include <ctype.h>

/* Euclidean distance calculation */
__host__  __device__ long distD(int x,int y,int N,long*dt)
{
	int id;
	if(x>y)
	{
		x=x+y;y=x-y;x=x-y;
	}
	id=x*(N-1)+(y-1)-(x*(x+1)/2);
	return(dt[id]);
}
/*A kenel function that finds a minimal weighted neighbor using TPN mapping strategy*/
__global__ void tsp(int *rt,long cost,unsigned long long *dst_tid,long cit,long *dt,long sol)
{

	long i,j;
	long change=0;
	long id=threadIdx.x+blockIdx.x*blockDim.x;
	if(id<sol)
	{
		
		i=cit-2-floorf(((int)__dsqrt_rn(8*(sol-id-1)+1)-1)/2);
		j=id-i*(cit-1)+(i*(i+1)/2)+1;
		change=distD(rt[i],rt[j],cit,dt)+distD(rt[(i+1)%cit],rt[(j+1)%cit],cit,dt)
			-distD(rt[i],rt[(i+1)%cit],cit,dt)-distD(rt[j],rt[(j+1)%cit],cit,dt);
		cost+=change;	
		if(change < 0)
			 atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
		
	}
	
}
/* At each IHC steps, XY coordinates are arranged using next initial solution's order*/
void twoOpt(int x,int y,int *route,int city)
{
	int *tmp_r;
	int i,j;
	tmp_r=(int*)malloc(sizeof(int)*(y-x));	
	for(j=0,i=y;i>x;i--,j++)
	{
		tmp_r[j]=route[i];
	}
	for(j=0,i=x+1;i<=y;i++,j++)
	{
		route[i]=tmp_r[j];
	}
	free(tmp_r);
}

int main(int argc, char *argv[])
{
	int ch, cnt, in1;
	float in2, in3;
	FILE *f;
	float *posx, *posy;
	char str[256];  
	long dst,d,tid,x,y, cities;
        unsigned long long *d_dst_tid;
	
	int blk,thrd;
	clock_t start,end;
	long sol;
	int *r,i,j;
	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}

	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

	ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
	fscanf(f, "%s\n", str);
	cities = atoi(str);
	if (cities <= 2) {fprintf(stderr, "only %d cities\n", cities);  exit(-1);}

	sol=cities*(cities-1)/2;
	posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	r = (int *)malloc(sizeof(int) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	fscanf(f, "%s\n", str);
	if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

	cnt = 0;

	while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) 
	{
		posx[cnt] = in2;
		posy[cnt] = in3;
		cnt++;
		if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
		if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	}

	if (cnt != cities) {fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);  exit(-1);}
	fscanf(f, "%s", str);
	if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}
    	fflush(f);
	fclose(f);
	/*Distance matrix */
	long *dist_mat=(long*)malloc(sizeof(long)*sol);
	int k=0;	
	for (int i = 0; i < cities; ++i)
	{
		for (int j = i+1; j < cities; ++j)
		{
		dist_mat[k] = sqrtf(pow(posx[i] - posx[j], 2)
		             +powf(posy[i] - posy[j], 2));
		k++;		
		}
	}
	/* CUDA threads and block configuration */
	if(sol < 1024)
	{
		blk = 1;
		thrd = cities;
	}
	else
	{
	blk=(sol-1)/1024+1;
	thrd=1024;
	}
	/*Initial solution construction using NN approach*/
	r[0]=0;
	k=1;i=0;float min;int minj,mini,count=1,flag=0;dst=0;
	int *v=(int*)calloc(cities,sizeof(int));
	v[0]=1;
	while(count!=cities)
	{
		flag=0;
		for(j=1;j<cities;j++)
		{
			if(i!=j && !v[j])
			{
				int id;
				if(i>j)
					id=j*(cities-1)+(i-1)-(j*(j+1)/2);
				else
					id=i*(cities-1)+(j-1)-(i*(i+1)/2);
	
				min=dist_mat[id];
				minj=j;
				break;	
			}
		}

		for(j=minj+1;j<cities;j++)
		{
			 if( !v[j])
			{
				int id;
				if(i>j)
					id=j*(cities-1)+(i-1)-(j*(j+1)/2);
				else
					id=i*(cities-1)+(j-1)-(i*(i+1)/2);	
				if(min>dist_mat[id])
				{
					min=dist_mat[id];
					mini=j;
					flag=1;				
				}
			}
		}
		if(flag==0)
			i=minj;
		else
			i=mini;
		dst+=min;
		r[k++]=i;v[i]=1;
		count++;
	}
	free(v);
	j=r[cities-1];
	i=0;
	int id=i*(cities-1)+(j-1)-(i*(i+1)/2);
	dst+=dist_mat[id];
	count=1;
	start = clock();
	hipEvent_t strt, stp;
	hipEventCreate(&strt);
	hipEventCreate(&stp);
 	unsigned long long dst_tid = (((long)dst+1) << 32) -1;
        unsigned long long dtid;
	int *d_r;
    	long *d_mt;
	printf("cities : %ld\ninitial cost : %ld\n",cities,dst);

	if(hipSuccess!=hipMalloc((void**)&d_dst_tid,sizeof(unsigned long long)))printf("\nAllocating memory for dst_tid on GPU");
    	if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))printf("\ntransfer on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_mt,sizeof(long)*sol))printf("\nAllocating memory for thread id on GPU");
    	if(hipSuccess!=hipMalloc((void**)&d_r,sizeof(int)*cities))printf("\nAllocating memory for thread id on GPU");
	if(hipSuccess!=hipMemcpy(d_mt,dist_mat,sizeof(long)*(sol),hipMemcpyHostToDevice))printf("\ntransfer on GPU 1");
    	if(hipSuccess!=hipMemcpy(d_r,r,sizeof(int)*cities,hipMemcpyHostToDevice))printf("\ntransfer on GPU 1");

	tsp<<<blk,thrd>>>(d_r,dst,d_dst_tid,cities,d_mt,sol);

	if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
	printf("\nCan't transfer minimal cost back to CPU");

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, strt, stp);
  	d = dtid >> 32;
	printf("\nfirst cost found %ld",d);	
	while( d < dst )
	{
		dst=d;
		tid = dtid & ((1ull<<32)-1); 
		x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
		y=tid-x*(cities-1)+(x*(x+1)/2)+1;
		twoOpt(x,y,r,cities);
		unsigned long long dst_tid = (((long)dst+1) << 32) -1;
    		if(hipSuccess!=hipMemcpy(d_r,r,sizeof(int)*cities,hipMemcpyHostToDevice))printf("\ntransfer on GPU 1");
    	        if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
		printf("\ntransfer on GPU");

		tsp<<<blk,thrd>>>(d_r,dst,d_dst_tid,cities,d_mt,sol);
		if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
		printf("\nCan't transfer minimal cost back to CPU");
	  	d = dtid >> 32;
		count++;
	}
	printf("\nMinimal Distance : %ld\n",d);

	printf("\nnumber of time climbed %d\n",count);
	end = clock();
	double t=((double) (end - start)) / CLOCKS_PER_SEC;
	printf("\ntime : %f\n",t);

	hipFree(d_r);
	hipFree(d_mt);
	hipFree(d_dst_tid);
	free(posx);
	free(posy);
	free(r);
	return 0;
}
